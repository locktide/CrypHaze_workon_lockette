#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/


/**
 * @section DESCRIPTION
 *
 * This file implements MD5 multihash cracking.
 */

#include <stdint.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

//#include "CUDA_Common/cuPrintf.cu"

#include "MFN_CUDA_device/MFN_CUDA_incrementors.h"
#include "MFN_CUDA_device/MFN_CUDA_Common.h"
#include "MFN_CUDA_device/MFN_CUDA_MD5.h"

#if !defined(__HIPCC__)
    // define the keywords, so that the IDE does not complain about them
    #define __global__
    #define __device__
    #define __shared__
    #define __constant__
    #define blockIdx.x 1
    #define blockDim.x 1
    #define threadIdx.x 1
    #define __align__() /**/
#endif

/**
 * The maximum password length supported by this hash type.
 */
#define MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_PASSLEN 48

/**
 * The maximum charset length supported by this hash type.
 */
#define MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH 128


// Define the constant types used by the kernels here.
__device__ __constant__ __align__(16) uint8_t deviceCharsetPlainMD5[MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH * \
    MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_PASSLEN];
__device__ __constant__ __align__(16) uint8_t deviceReverseCharsetPlainMD5[MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH * \
    MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_PASSLEN];
__device__ __constant__ uint8_t charsetLengthsPlainMD5[MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_PASSLEN];
__device__ __constant__ __align__(16) uint8_t  constantBitmapAPlainMD5[8192];

/**
 * Constant parameters go here instead of getting passed as kernel arguments.
 * This allows for faster accesses (as they are cached, and all threads will
 * be accessing the same element), and also reduces the shared memory usage,
 * which may allow for better occupancy in the future.  The kernels will load
 * these as needed, and theoretically will not need registers for some of them,
 * which will help reduce the register pressure on kernels.  Hopefully.
 */

// Password length.  Needed for some offset calculations.
__device__ __constant__ uint8_t passwordLengthPlainMD5;

// Number of hashes present in memory.
__device__ __constant__ uint64_t numberOfHashesPlainMD5;

// Address of the hashlist in global memory.
__device__ __constant__ uint8_t *deviceGlobalHashlistAddressPlainMD5;

// Addresses of the various global bitmaps.
__device__ __constant__ uint8_t *deviceGlobalBitmapAPlainMD5;
__device__ __constant__ uint8_t *deviceGlobalBitmapBPlainMD5;
__device__ __constant__ uint8_t *deviceGlobalBitmapCPlainMD5;
__device__ __constant__ uint8_t *deviceGlobalBitmapDPlainMD5;

// Addresses of the arrays for found passwords & success flags
__device__ __constant__ uint8_t *deviceGlobalFoundPasswordsPlainMD5;
__device__ __constant__ uint8_t *deviceGlobalFoundPasswordFlagsPlainMD5;

__device__ __constant__ uint8_t *deviceGlobalStartPointsPlainMD5;
__device__ __constant__ uint32_t *deviceGlobalStartPasswords32PlainMD5;

__device__ __constant__ uint32_t deviceNumberStepsToRunPlainMD5;
__device__ __constant__ uint64_t deviceNumberThreadsPlainMD5;





// Defined if we are using the loadPasswords32/storePasswords32
#define USE_NEW_PASSWORD_LOADING 1


#define MAKE_MFN_MD5_KERNEL1_8LENGTH(pass_len) \
__global__ void MFNHashTypePlainCUDA_MD5_GeneratedKernel_##pass_len () { \
    uint32_t b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d; \
    uint32_t password_count = 0, passOffset; \
    __shared__ uint8_t __align__(16) sharedCharsetPlainMD5[MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH * pass_len]; \
    __shared__ uint8_t __align__(16) sharedReverseCharsetPlainMD5[MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH * pass_len]; \
    __shared__ uint8_t sharedCharsetLengthsPlainMD5[pass_len]; \
    __shared__ uint8_t __align__(16) sharedBitmap[8192]; \
    if (threadIdx.x == 0) { \
        uint64_t *sharedCharset64 = (uint64_t *)sharedCharsetPlainMD5; \
        uint64_t *deviceCharset64 = (uint64_t *)deviceCharsetPlainMD5; \
        uint64_t *sharedReverseCharset64 = (uint64_t *)sharedReverseCharsetPlainMD5; \
        uint64_t *deviceReverseCharset64 = (uint64_t *)deviceReverseCharsetPlainMD5; \
        uint64_t *constantBitmap64 = (uint64_t *)constantBitmapAPlainMD5; \
        uint64_t *sharedBitmap64 = (uint64_t *)sharedBitmap; \
        for (a = 0; a < ((MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH * pass_len) / 8); a++) { \
            sharedCharset64[a] = deviceCharset64[a]; \
            sharedReverseCharset64[a] = deviceReverseCharset64[a]; \
        } \
        for (a = 0; a < pass_len; a++) { \
            sharedCharsetLengthsPlainMD5[a] = charsetLengthsPlainMD5[a]; \
        } \
        for (a = 0; a < 8192 / 8; a++) { \
            sharedBitmap64[a] = constantBitmap64[a]; \
        } \
    } \
    syncthreads(); \
    b0 = b1 = b2 = b3 = b4 = b5 = b6 = b7 = b8 = b9 = b10 = b11 = b12 = b13 = b14 = b15 = 0; \
    b14 = pass_len * 8; \
    if (USE_NEW_PASSWORD_LOADING) { \
        loadPasswords32(deviceGlobalStartPasswords32PlainMD5, deviceNumberThreadsPlainMD5, pass_len); \
    } else {\
        if (charsetLengthsPlainMD5[1] == 0) { \
            loadPasswordSingle(sharedCharsetPlainMD5, deviceGlobalStartPointsPlainMD5, deviceNumberThreadsPlainMD5, pass_len); \
        } else { \
            loadPasswordMultiple(sharedCharsetPlainMD5, deviceGlobalStartPointsPlainMD5, deviceNumberThreadsPlainMD5, pass_len, MFN_HASH_TYPE_PLAIN_CUDA_MD5_MAX_CHARSET_LENGTH); \
        } \
        ResetCharacterAtPosition(0x80, pass_len, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
    } \
    while (password_count < deviceNumberStepsToRunPlainMD5) { \
        MD5_FIRST_3_ROUNDS(); \
        if (pass_len <= 8) { \
            MD5II (a, b, c, d, b0, MD5S41, 0xf4292244); \
            MD5II (d, a, b, c, b7, MD5S42, 0x432aff97); \
            MD5II (c, d, a, b, b14, MD5S43, 0xab9423a7); \
            MD5II (b, c, d, a, b5, MD5S44, 0xfc93a039); \
            MD5II (a, b, c, d, b12, MD5S41, 0x655b59c3); \
            if ((sharedBitmap[(a & 0x0000ffff) >> 3] >> (a & 0x00000007)) & 0x00000001) { \
                if (!(deviceGlobalBitmapAPlainMD5) || ((deviceGlobalBitmapAPlainMD5[(a >> 3) & 0x07FFFFFF] >> (a & 0x7)) & 0x1)) { \
                    MD5II (d, a, b, c, b3, MD5S42, 0x8f0ccc92); \
                    if (!deviceGlobalBitmapDPlainMD5 || ((deviceGlobalBitmapDPlainMD5[(d >> 3) & 0x07FFFFFF] >> (d & 0x7)) & 0x1)) { \
                        MD5II (c, d, a, b, b10, MD5S43, 0xffeff47d); \
                        if (!deviceGlobalBitmapCPlainMD5 || ((deviceGlobalBitmapCPlainMD5[(c >> 3) & 0x07FFFFFF] >> (c & 0x7)) & 0x1)) { \
                            MD5II (b, c, d, a, b1, MD5S44, 0x85845dd1); \
                            if (!deviceGlobalBitmapBPlainMD5 || ((deviceGlobalBitmapBPlainMD5[(b >> 3) & 0x07FFFFFF] >> (b & 0x7)) & 0x1)) { \
                                checkHashList128LE(a, b, c, d, b0, b1, b2, b3, \
                                    deviceGlobalFoundPasswordsPlainMD5, deviceGlobalFoundPasswordFlagsPlainMD5, \
                                    deviceGlobalHashlistAddressPlainMD5, numberOfHashesPlainMD5, \
                                    passwordLengthPlainMD5); \
            }   }   }   }   } \
        } else if (pass_len > 8) { \
            MD5II (a, b, c, d, b0, MD5S41, 0xf4292244); \
            MD5II (d, a, b, c, b7, MD5S42, 0x432aff97); \
            MD5II (c, d, a, b, b14, MD5S43, 0xab9423a7); \
            MD5II (b, c, d, a, b5, MD5S44, 0xfc93a039); \
            MD5II (a, b, c, d, b12, MD5S41, 0x655b59c3); \
            MD5II (d, a, b, c, b3, MD5S42, 0x8f0ccc92); \
            MD5II (c, d, a, b, b10, MD5S43, 0xffeff47d); \
            MD5II (b, c, d, a, b1, MD5S44, 0x85845dd1); \
            MD5II (a, b, c, d, b8, MD5S41, 0x6fa87e4f); \
            MD5II (d, a, b, c, b15, MD5S42, 0xfe2ce6e0); \
            MD5II (c, d, a, b, b6, MD5S43, 0xa3014314); \
            MD5II (b, c, d, a, b13, MD5S44, 0x4e0811a1); \
            MD5II (a, b, c, d, b4, MD5S41, 0xf7537e82);  \
            if ((sharedBitmap[(a & 0x0000ffff) >> 3] >> (a & 0x00000007)) & 0x00000001) { \
                if (!(deviceGlobalBitmapAPlainMD5) || ((deviceGlobalBitmapAPlainMD5[(a >> 3) & 0x07FFFFFF] >> (a & 0x7)) & 0x1)) { \
                    MD5II (d, a, b, c, b11, MD5S42, 0xbd3af235); \
                    if (!deviceGlobalBitmapDPlainMD5 || ((deviceGlobalBitmapDPlainMD5[(d >> 3) & 0x07FFFFFF] >> (d & 0x7)) & 0x1)) { \
                        MD5II (c, d, a, b, b2, MD5S43, 0x2ad7d2bb);  \
                        if (!deviceGlobalBitmapCPlainMD5 || ((deviceGlobalBitmapCPlainMD5[(c >> 3) & 0x07FFFFFF] >> (c & 0x7)) & 0x1)) { \
                            MD5II (b, c, d, a, b9, MD5S44, 0xeb86d391); \
                            if (!deviceGlobalBitmapBPlainMD5 || ((deviceGlobalBitmapBPlainMD5[(b >> 3) & 0x07FFFFFF] >> (b & 0x7)) & 0x1)) { \
                            checkHashList128LE(a, b, c, d, b0, b1, b2, b3, \
                                    deviceGlobalFoundPasswordsPlainMD5, deviceGlobalFoundPasswordFlagsPlainMD5, \
                                    deviceGlobalHashlistAddressPlainMD5, numberOfHashesPlainMD5, \
                                    passwordLengthPlainMD5); \
            }   }   }   }   }\
        } \
        if (charsetLengthsPlainMD5[1] == 0) { \
                makeMFNSingleIncrementors##pass_len (sharedCharsetPlainMD5, sharedReverseCharsetPlainMD5, sharedCharsetLengthsPlainMD5); \
        } else { \
                makeMFNMultipleIncrementors##pass_len (sharedCharsetPlainMD5, sharedReverseCharsetPlainMD5, sharedCharsetLengthsPlainMD5); \
        } \
        password_count++; \
    } \
    if (USE_NEW_PASSWORD_LOADING) { \
        storePasswords32(deviceGlobalStartPasswords32PlainMD5, deviceNumberThreadsPlainMD5, pass_len); \
    } \
}

MAKE_MFN_MD5_KERNEL1_8LENGTH(1);
MAKE_MFN_MD5_KERNEL1_8LENGTH(2);
MAKE_MFN_MD5_KERNEL1_8LENGTH(3);
MAKE_MFN_MD5_KERNEL1_8LENGTH(4);
MAKE_MFN_MD5_KERNEL1_8LENGTH(5);
MAKE_MFN_MD5_KERNEL1_8LENGTH(6);
MAKE_MFN_MD5_KERNEL1_8LENGTH(7);
MAKE_MFN_MD5_KERNEL1_8LENGTH(8);
MAKE_MFN_MD5_KERNEL1_8LENGTH(9);
MAKE_MFN_MD5_KERNEL1_8LENGTH(10);
MAKE_MFN_MD5_KERNEL1_8LENGTH(11);
MAKE_MFN_MD5_KERNEL1_8LENGTH(12);
MAKE_MFN_MD5_KERNEL1_8LENGTH(13);
MAKE_MFN_MD5_KERNEL1_8LENGTH(14);
MAKE_MFN_MD5_KERNEL1_8LENGTH(15);
MAKE_MFN_MD5_KERNEL1_8LENGTH(16);

extern "C" hipError_t MFNHashTypePlainCUDA_MD5_CopyValueToConstant(
        const char *symbolName, void *hostDataAddress, size_t bytesToCopy) {
    return hipMemcpyToSymbol(HIP_SYMBOL(symbolName), hostDataAddress, bytesToCopy);
}

extern "C" hipError_t MFNHashTypePlainCUDA_MD5_LaunchKernel(uint32_t passwordLength, uint32_t Blocks, uint32_t Threads) {
    //printf("MFNHashTypePlainCUDA_MD5_LaunchKernel()\n");

    //cudaPrintfInit();
    switch (passwordLength) {
        case 1:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_1 <<< Blocks, Threads >>> ();
            break;
        case 2:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_2 <<< Blocks, Threads >>> ();
            break;
        case 3:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_3 <<< Blocks, Threads >>> ();
            break;
        case 4:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_4 <<< Blocks, Threads >>> ();
            break;
        case 5:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_5 <<< Blocks, Threads >>> ();
            break;
        case 6:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_6 <<< Blocks, Threads >>> ();
            break;
        case 7:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_7 <<< Blocks, Threads >>> ();
            break;
        case 8:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_8 <<< Blocks, Threads >>> ();
            break;
        case 9:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_9 <<< Blocks, Threads >>> ();
            break;
        case 10:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_10 <<< Blocks, Threads >>> ();
            break;
        case 11:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_11 <<< Blocks, Threads >>> ();
            break;
        case 12:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_12 <<< Blocks, Threads >>> ();
            break;
        case 13:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_13 <<< Blocks, Threads >>> ();
            break;
        case 14:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_14 <<< Blocks, Threads >>> ();
            break;
        case 15:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_15 <<< Blocks, Threads >>> ();
            break;
        case 16:
            MFNHashTypePlainCUDA_MD5_GeneratedKernel_16 <<< Blocks, Threads >>> ();
            break;
        default:
            printf("Password length %d unsupported!\n", passwordLength);
            exit(1);
            break;
            
    }
    //cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();

    return hipGetLastError();
}
