#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/


/**
 * @section DESCRIPTION
 *
 * This file implements NTLM multihash cracking.
 */

#include <stdint.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "MFN_CUDA_device/MFN_CUDA_NTLM_incrementors.h"
#include "MFN_CUDA_device/MFN_CUDA_Common.h"
#include "MFN_CUDA_device/MFN_CUDA_MD4.h"

#if !defined(__HIPCC__)
    // define the keywords, so that the IDE does not complain about them
    #define __global__
    #define __device__
    #define __shared__
    #define __constant__
    #define blockIdx.x 1
    #define blockDim.x 1
    #define threadIdx.x 1
    #define __align__() /**/
#endif

/**
 * The maximum password length supported by this hash type.
 */
#define MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_PASSLEN 28

/**
 * The maximum charset length supported by this hash type.
 */
#define MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH 128


// Define the constant types used by the kernels here.
__device__ __constant__ __align__(16) uint8_t deviceCharsetPlainNTLM[MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH * \
    MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_PASSLEN];
__device__ __constant__ __align__(16) uint8_t deviceReverseCharsetPlainNTLM[MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH * \
    MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_PASSLEN];
__device__ __constant__ uint8_t charsetLengthsPlainNTLM[MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_PASSLEN];
__device__ __constant__ __align__(16) uint8_t  constantBitmapAPlainNTLM[8192];

/**
 * Constant parameters go here instead of getting passed as kernel arguments.
 * This allows for faster accesses (as they are cached, and all threads will
 * be accessing the same element), and also reduces the shared memory usage,
 * which may allow for better occupancy in the future.  The kernels will load
 * these as needed, and theoretically will not need registers for some of them,
 * which will help reduce the register pressure on kernels.  Hopefully.
 */

// Password length.  Needed for some offset calculations.
__device__ __constant__ uint8_t passwordLengthPlainNTLM;

// Number of hashes present in memory.
__device__ __constant__ uint64_t numberOfHashesPlainNTLM;

// Address of the hashlist in global memory.
__device__ __constant__ uint8_t *deviceGlobalHashlistAddressPlainNTLM;

// Addresses of the various global bitmaps.
__device__ __constant__ uint8_t *deviceGlobalBitmapAPlainNTLM;
__device__ __constant__ uint8_t *deviceGlobalBitmapBPlainNTLM;
__device__ __constant__ uint8_t *deviceGlobalBitmapCPlainNTLM;
__device__ __constant__ uint8_t *deviceGlobalBitmapDPlainNTLM;

// Addresses of the arrays for found passwords & success flags
__device__ __constant__ uint8_t *deviceGlobalFoundPasswordsPlainNTLM;
__device__ __constant__ uint8_t *deviceGlobalFoundPasswordFlagsPlainNTLM;

__device__ __constant__ uint8_t *deviceGlobalStartPointsPlainNTLM;
__device__ __constant__ uint32_t *deviceGlobalStartPasswords32PlainNTLM;

__device__ __constant__ uint32_t deviceNumberStepsToRunPlainNTLM;
__device__ __constant__ uint64_t deviceNumberThreadsPlainNTLM;


/**
 * The loadPassword32 and storePassword32 methods are the preferred method for loading plains.
 * 
 * These work by loading the b0,b1,b2, etc directly from the memory space
 * as plaintext passwords.  At the end of each kernel execution, the current
 * passwords are stored back to the array.  This prevents the need to transfer
 * more plain start points to each thread when the kernel starts again.
 * 
 * @param pa Password initial array
 * @param dt Device number threads
 * @param pl Password length
 */
#define loadNTLMPasswords32(pa, dt, pl) { \
a = thread_index; \
b = pa[a]; \
b0 = (b & 0xff) | ((b & 0xff00) << 8); \
if (pl > 1) {b1 = ((b & 0xff0000) >> 16) | ((b & 0xff000000) >> 8);} \
if (pl > 3) {a += dt; b = pa[a]; b2 = (b & 0xff) | ((b & 0xff00) << 8);} \
if (pl > 5) {b3 = ((b & 0xff0000) >> 16) | ((b & 0xff000000) >> 8);} \
if (pl > 7) {a += dt; b = pa[a]; b4 = (b & 0xff) | ((b & 0xff00) << 8);} \
if (pl > 9) {b5 = ((b & 0xff0000) >> 16) | ((b & 0xff000000) >> 8);} \
if (pl > 11) {a += dt; b6 = pa[a]; b6 = (b & 0xff) | ((b & 0xff00) << 8);} \
if (pl > 13) {b7 = ((b & 0xff0000) >> 16) | ((b & 0xff000000) >> 8);} \
}

#define storeNTLMPasswords32(pa, dt, pl) { \
b = (b0 & 0xff) | ((b0 & 0xff0000) >> 8); \
if (pl > 1) {b |= (b1 & 0xff) << 16 | ((b1 & 0xff0000) << 8);} \
pa[thread_index + 0] = b; \
if (pl > 3) {b = (b2 & 0xff) | ((b2 & 0xff0000) >> 8);} \
if (pl > 5) {b |= (b3 & 0xff) << 16 | ((b3 & 0xff0000) << 8);} \
if (pl > 3) {pa[thread_index + (dt * 1)] = b;} \
if (pl > 7) {b = (b4 & 0xff) | ((b4 & 0xff0000) >> 8);} \
if (pl > 9) {b |= (b5 & 0xff) << 16 | ((b5 & 0xff0000) << 8);} \
if (pl > 7) {pa[thread_index + (dt * 2)] = b;} \
if (pl > 11) {b = (b6 & 0xff) | ((b6 & 0xff0000) >> 8);} \
if (pl > 13) {b |= (b7 & 0xff) << 16 | ((b7 & 0xff0000) << 8);} \
if (pl > 11) {pa[thread_index + (dt * 3)] = b;} \
}


/**
 * Searches for a 128 bit little endian NTLM hash in the global memory.
 *
 * This function takes the calculated hash values (a, b, c, d), the password
 * in b0, b1, etc (as NTLM style!), and the various global memory pointers
 * and searches for the hash.  If it is found, it reports it in the appropriate
 * method.
 *
 * @param a,b,c,d The calculated hash values.
 * @param b0-b7 The registers containing the input block in NTLM format
 * @param sharedBitmapA The address of the 8kb bitmap ideally in shared memory
 * @param deviceGlobalBitmap{A,B,C,D} The addresses (or null) of the device global bitmaps.
 * @param deviceGlobalFoundPasswords The address of the found-password array
 * @param deviceGlobalFoundPasswordFlags The address of the found-password flag array
 * @param deviceGlobalHashlistAddress The address of the 128-bit hash global hashlist
 * @param numberOfHashes The number of hashes being searched for currently
 * @param passwordLength The current password length
 */
__device__ inline void checkHash128LENTLM(uint32_t &a, uint32_t &b, uint32_t &c, uint32_t &d,
        uint32_t &b0, uint32_t &b1, uint32_t &b2, uint32_t &b3, 
        uint32_t &b4, uint32_t &b5, uint32_t &b6, uint32_t &b7, 
        uint8_t *sharedBitmapA,
        uint8_t *deviceGlobalBitmapA, uint8_t *deviceGlobalBitmapB,
        uint8_t *deviceGlobalBitmapC, uint8_t *deviceGlobalBitmapD,
        uint8_t *deviceGlobalFoundPasswords, uint8_t *deviceGlobalFoundPasswordFlags,
        uint8_t *deviceGlobalHashlistAddress, uint64_t numberOfHashes,
        uint8_t passwordLength) {
    if ((sharedBitmapA[(a & 0x0000ffff) >> 3] >> (a & 0x00000007)) & 0x00000001) {
        if (!(deviceGlobalBitmapA) || ((deviceGlobalBitmapA[(a >> 3) & 0x07FFFFFF] >> (a & 0x7)) & 0x1)) {
            if (!deviceGlobalBitmapB || ((deviceGlobalBitmapB[(b >> 3) & 0x07FFFFFF] >> (b & 0x7)) & 0x1)) {
                if (!deviceGlobalBitmapC || ((deviceGlobalBitmapC[(c >> 3) & 0x07FFFFFF] >> (c & 0x7)) & 0x1)) {
                    if (!deviceGlobalBitmapD || ((deviceGlobalBitmapD[(d >> 3) & 0x07FFFFFF] >> (d & 0x7)) & 0x1)) {
                        uint32_t search_high, search_low, search_index, current_hash_value;
                        uint32_t *DEVICE_Hashes_32 = (uint32_t *) deviceGlobalHashlistAddress;
                        search_high = numberOfHashes;
                        search_low = 0;
                        while (search_low < search_high) {
                            // Midpoint between search_high and search_low
                            search_index = search_low + (search_high - search_low) / 2;
                            current_hash_value = DEVICE_Hashes_32[4 * search_index];
                            // Adjust search_high & search_low to work through space
                            if (current_hash_value < a) {
                                search_low = search_index + 1;
                            } else {
                                search_high = search_index;
                            }
                            if ((a == current_hash_value) && (search_low < numberOfHashes)) {
                                // Break out of the search loop - search_index is on a value
                                break;
                            }
                        }
                        // Broke out of the while loop

                        // If the loaded value does not match, there are no matches - just return.
                        if (a != current_hash_value) {
                            return;
                        }
                        // We've broken out of the loop, search_index should be on a matching value
                        // Loop while the search index is the same - linear search through this to find all possible
                        // matching passwords.
                        // We first need to move backwards to the beginning, as we may be in the middle of a set of matching hashes.
                        // If we are index 0, do NOT subtract, as we will wrap and this goes poorly.

                        while (search_index && (a == DEVICE_Hashes_32[(search_index - 1) * 4])) {
                            search_index--;
                        }
                        while ((a == DEVICE_Hashes_32[search_index * 4])) {
                            if (b == DEVICE_Hashes_32[search_index * 4 + 1]) {
                                if (c == DEVICE_Hashes_32[search_index * 4 + 2]) {
                                    if (d == DEVICE_Hashes_32[search_index * 4 + 3]) {
                                        // Copy the password to the correct location.
                                        switch (passwordLength) {
                                            case 16:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 15] = (b7 >> 16) & 0xff;
                                            case 15:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 14] = (b7 >> 0) & 0xff;
                                            case 14:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 13] = (b6 >> 16) & 0xff;
                                            case 13:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 12] = (b6 >> 0) & 0xff;
                                            case 12:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 11] = (b5 >> 16) & 0xff;
                                            case 11:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 10] = (b5 >> 0) & 0xff;
                                            case 10:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 9] = (b4 >> 16) & 0xff;
                                            case 9:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 8] = (b4 >> 0) & 0xff;
                                            case 8:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 7] = (b3 >> 16) & 0xff;
                                            case 7:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 6] = (b3 >> 0) & 0xff;
                                            case 6:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 5] = (b2 >> 16) & 0xff;
                                            case 5:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 4] = (b2 >> 0) & 0xff;
                                            case 4:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 3] = (b1 >> 16) & 0xff;
                                            case 3:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 2] = (b1 >> 0) & 0xff;
                                            case 2:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 1] = (b0 >> 16) & 0xff;
                                            case 1:
                                                deviceGlobalFoundPasswords[search_index * passwordLength + 0] = (b0 >> 0) & 0xff;
                                        }
                                        deviceGlobalFoundPasswordFlags[search_index] = (unsigned char) 1;
                                    }
                                }
                            }
                            search_index++;
                        }
                    }
                }
            }
        }
    }
}


#define MAKE_MFN_NTLM_KERNEL1_8LENGTH(pass_len) \
__global__ void MFNHashTypePlainCUDA_NTLM_GeneratedKernel_##pass_len () { \
    uint32_t b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d; \
    uint32_t password_count = 0, passOffset; \
    __shared__ uint8_t __align__(16) sharedCharsetPlainNTLM[MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH * pass_len]; \
    __shared__ uint8_t __align__(16) sharedReverseCharsetPlainNTLM[MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH * pass_len]; \
    __shared__ uint8_t __align__(16) sharedCharsetLengthsPlainNTLM[pass_len]; \
    __shared__ uint8_t __align__(16) sharedBitmap[8192]; \
    if (threadIdx.x == 0) { \
        uint64_t *sharedCharset64 = (uint64_t *)sharedCharsetPlainNTLM; \
        uint64_t *deviceCharset64 = (uint64_t *)deviceCharsetPlainNTLM; \
        uint64_t *sharedReverseCharset64 = (uint64_t *)sharedReverseCharsetPlainNTLM; \
        uint64_t *deviceReverseCharset64 = (uint64_t *)deviceReverseCharsetPlainNTLM; \
        uint64_t *constantBitmap64 = (uint64_t *)constantBitmapAPlainNTLM; \
        uint64_t *sharedBitmap64 = (uint64_t *)sharedBitmap; \
        for (a = 0; a < ((MFN_HASH_TYPE_PLAIN_CUDA_NTLM_MAX_CHARSET_LENGTH * pass_len) / 8); a++) { \
            sharedCharset64[a] = deviceCharset64[a]; \
            sharedReverseCharset64[a] = deviceReverseCharset64[a]; \
        } \
        for (a = 0; a < pass_len; a++) { \
            sharedCharsetLengthsPlainNTLM[a] = charsetLengthsPlainNTLM[a]; \
        } \
        for (a = 0; a < 8192 / 8; a++) { \
            sharedBitmap64[a] = constantBitmap64[a]; \
        } \
    } \
    syncthreads(); \
    b0 = b1 = b2 = b3 = b4 = b5 = b6 = b7 = b8 = b9 = b10 = b11 = b12 = b13 = b14 = b15 = 0; \
    b14 = pass_len * 16; \
    loadNTLMPasswords32(deviceGlobalStartPasswords32PlainNTLM, deviceNumberThreadsPlainNTLM, pass_len); \
    while (password_count < deviceNumberStepsToRunPlainNTLM) { \
        MD4_FIRST_2_ROUNDS(); \
        MD4HH (a, b, c, d, b0, MD4S31); \
        MD4HH (d, a, b, c, b8, MD4S32); \
        MD4HH (c, d, a, b, b4, MD4S33); \
        MD4HH (b, c, d, a, b12, MD4S34); \
        MD4HH (a, b, c, d, b2, MD4S31); \
        MD4HH (d, a, b, c, b10, MD4S32); \
        MD4HH (c, d, a, b, b6, MD4S33); \
        MD4HH (b, c, d, a, b14, MD4S34); \
        MD4HH (a, b, c, d, b1, MD4S31); \
        if (pass_len > 6) { \
            MD4HH (d, a, b, c, b9, MD4S32); \
            MD4HH (c, d, a, b, b5, MD4S33); \
            MD4HH (b, c, d, a, b13, MD4S34); \
            MD4HH (a, b, c, d, b3, MD4S31); \
            if (pass_len > 14) { \
                MD4HH (d, a, b, c, b11, MD4S32); \
                MD4HH (c, d, a, b, b7, MD4S33); \
            } \
        } \
        checkHash128LENTLM(a, b, c, d, b0, b1, b2, b3, \
            b4, b5, b6, b7, sharedBitmap, \
            deviceGlobalBitmapAPlainNTLM, deviceGlobalBitmapBPlainNTLM, \
            deviceGlobalBitmapCPlainNTLM, deviceGlobalBitmapDPlainNTLM, \
            deviceGlobalFoundPasswordsPlainNTLM, deviceGlobalFoundPasswordFlagsPlainNTLM, \
            deviceGlobalHashlistAddressPlainNTLM, numberOfHashesPlainNTLM, \
            passwordLengthPlainNTLM); \
        if (charsetLengthsPlainNTLM[1] == 0) { \
                makeMFNSingleIncrementorsNTLM##pass_len (sharedCharsetPlainNTLM, sharedReverseCharsetPlainNTLM, sharedCharsetLengthsPlainNTLM); \
        } else { \
                makeMFNMultipleIncrementorsNTLM##pass_len (sharedCharsetPlainNTLM, sharedReverseCharsetPlainNTLM, sharedCharsetLengthsPlainNTLM); \
        } \
        password_count++; \
    } \
    storeNTLMPasswords32(deviceGlobalStartPasswords32PlainNTLM, deviceNumberThreadsPlainNTLM, pass_len); \
}


MAKE_MFN_NTLM_KERNEL1_8LENGTH(1);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(2);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(3);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(4);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(5);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(6);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(7);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(8);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(9);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(10);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(11);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(12);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(13);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(14);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(15);
MAKE_MFN_NTLM_KERNEL1_8LENGTH(16);

extern "C" hipError_t MFNHashTypePlainCUDA_NTLM_CopyValueToConstant(
        const char *symbolName, void *hostDataAddress, size_t bytesToCopy) {
    return hipMemcpyToSymbol(HIP_SYMBOL(symbolName), hostDataAddress, bytesToCopy);
}

extern "C" hipError_t MFNHashTypePlainCUDA_NTLM_LaunchKernel(uint32_t passwordLength, uint32_t Blocks, uint32_t Threads) {
    //printf("MFNHashTypePlainCUDA_NTLM_LaunchKernel()\n");
    
    //cudaPrintfInit();
//    hipError_t errbefore = hipGetLastError();
//    if( hipSuccess != errbefore)
//      {
//        printf("MFNHashTypePlainCUDA_NTLM Cuda errorbefore: %s.\n", hipGetErrorString( errbefore) );
//      } else {
//        printf("No error before\n");
//      }

    
    switch (passwordLength) {
        case 1:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_1 <<< Blocks, Threads >>> ();
            break;
        case 2:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_2 <<< Blocks, Threads >>> ();
            break;
        case 3:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_3 <<< Blocks, Threads >>> ();
            break;
        case 4:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_4 <<< Blocks, Threads >>> ();
            break;
        case 5:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_5 <<< Blocks, Threads >>> ();
            break;
        case 6:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_6 <<< Blocks, Threads >>> ();
            break;
        case 7:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_7 <<< Blocks, Threads >>> ();
            break;
        case 8:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_8 <<< Blocks, Threads >>> ();
            break;
        case 9:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_9 <<< Blocks, Threads >>> ();
            break;
        case 10:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_10 <<< Blocks, Threads >>> ();
            break;
        case 11:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_11 <<< Blocks, Threads >>> ();
            break;
        case 12:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_12 <<< Blocks, Threads >>> ();
            break;
        case 13:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_13 <<< Blocks, Threads >>> ();
            break;
        case 14:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_14 <<< Blocks, Threads >>> ();
            break;
        case 15:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_15 <<< Blocks, Threads >>> ();
            break;
        case 16:
            MFNHashTypePlainCUDA_NTLM_GeneratedKernel_16 <<< Blocks, Threads >>> ();
            break;
        default:
            printf("Password length %d unsupported!\n", passwordLength);
            exit(1);
            break;

    }
    //cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
      {
        printf("MFNHashTypePlainCUDA_NTLM Cuda error: %s.\n", hipGetErrorString( err) );
      }

    return err;
}
