#include "hip/hip_runtime.h"
/*
Cryptohaze GPU Rainbow Tables
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

// CUDA SHA1 kernels for table generation.

// This is here so Netbeans doesn't error-spam my IDE
#if !defined(__HIPCC__)
    // define the keywords, so that the IDE does not complain about them
    #define __global__
    #define __device__
    #define __shared__
    #define __constant__
    #define blockIdx.x 1
    #define blockDim.x 1
    #define threadIdx.x 1
#endif

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>
#ifdef _WIN32
#include "windows/stdint.h"
#else
#include <stdint.h>
#endif
typedef uint32_t uint32_t;


// Some CUDA variables
__device__ __constant__ unsigned char SHA1_Generate_Device_Charset_Constant[512]; // Constant space for charset
__device__ __constant__ uint32_t SHA1_Generate_Device_Charset_Length; // Character set length
__device__ __constant__ uint32_t SHA1_Generate_Device_Chain_Length; // May as well pull it from constant memory... faster.
__device__ __constant__ uint32_t SHA1_Generate_Device_Number_Of_Chains; // Same, may as well be constant.
__device__ __constant__ uint32_t SHA1_Generate_Device_Table_Index;
__device__ __constant__ uint32_t SHA1_Generate_Device_Number_Of_Threads; // It needs this, and can't easily calculate it


#include "../../inc/CUDA_Common/CUDA_SHA1.h"
#include "../../inc/CUDA_Common/Hash_Common.h"
#include "../../inc/GRT_CUDA_device/CUDA_Reduction_Functions.h"
#include "../../inc/GRT_CUDA_device/CUDA_Load_Store_Registers.h"


#define CREATE_SHA1_GEN_KERNEL(length) \
__global__ void MakeSHA1ChainLen##length(unsigned char *InitialPasswordArray, unsigned char *OutputHashArray, \
    uint32_t PasswordSpaceOffset, uint32_t StartChainIndex, uint32_t StepsToRun, uint32_t charset_offset) { \
    const int pass_length = length; \
    uint32_t CurrentStep, PassCount, password_index; \
    uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
    uint32_t a,b,c,d,e; \
    uint32_t *InitialArray32; \
    uint32_t *OutputArray32; \
    InitialArray32 = (uint32_t *)InitialPasswordArray; \
    OutputArray32 = (uint32_t *)OutputHashArray; \
    __shared__ char charset[512]; \
    copySingleCharsetToShared(charset, SHA1_Generate_Device_Charset_Constant); \
    password_index = ((blockIdx.x*blockDim.x + threadIdx.x) + (PasswordSpaceOffset * SHA1_Generate_Device_Number_Of_Threads)); \
    if (password_index >= SHA1_Generate_Device_Number_Of_Chains) { \
        return; \
    } \
    clearB0toB15(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15); \
    LoadMD5RegistersFromGlobalMemory(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15, \
        InitialArray32, SHA1_Generate_Device_Number_Of_Chains, password_index, pass_length); \
    for (PassCount = 0; PassCount < StepsToRun; PassCount++) { \
        CurrentStep = PassCount + StartChainIndex; \
        b15 = ((pass_length * 8) & 0xff) << 24 | (((pass_length * 8) >> 8) & 0xff) << 16; \
        SetCharacterAtPosition(0x80, pass_length, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15 ); \
        SHA_TRANSFORM(a, b, c, d, e, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
        a = reverse(a);b = reverse(b);c = reverse(c);d = reverse(d);e = reverse(e); \
        clearB0toB15(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15); \
        reduceSingleCharsetNormal(b0, b1, b2, a, b, c, d, CurrentStep, charset, charset_offset, pass_length, SHA1_Generate_Device_Table_Index); \
        charset_offset++; \
        if (charset_offset >= SHA1_Generate_Device_Charset_Length) { \
            charset_offset = 0; \
        } \
    } \
    if (CurrentStep >= (SHA1_Generate_Device_Chain_Length - 1)) { \
        OutputArray32[0 * SHA1_Generate_Device_Number_Of_Chains + password_index] = a; \
        OutputArray32[1 * SHA1_Generate_Device_Number_Of_Chains + password_index] = b; \
        OutputArray32[2 * SHA1_Generate_Device_Number_Of_Chains + password_index] = c; \
        OutputArray32[3 * SHA1_Generate_Device_Number_Of_Chains + password_index] = d; \
        OutputArray32[4 * SHA1_Generate_Device_Number_Of_Chains + password_index] = e; \
    } \
    else { \
    SaveMD5RegistersIntoGlobalMemory(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15, \
        InitialArray32, SHA1_Generate_Device_Number_Of_Chains, password_index, pass_length); \
    } \
}




CREATE_SHA1_GEN_KERNEL(6)
CREATE_SHA1_GEN_KERNEL(7)
CREATE_SHA1_GEN_KERNEL(8)
CREATE_SHA1_GEN_KERNEL(9)
CREATE_SHA1_GEN_KERNEL(10)




extern "C" void copyConstantsToSHA1(unsigned char *HOST_Charset, uint32_t HOST_Charset_Length,
    uint32_t HOST_Chain_Length, uint32_t HOST_Number_Of_Chains, uint32_t HOST_Table_Index,
    uint32_t HOST_Number_Of_Threads) {

    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Charset_Constant"),HOST_Charset, 512);
    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Charset_Length"), &HOST_Charset_Length, sizeof(uint32_t));

    // Copy general table parameters to constant space
    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Chain_Length"), &HOST_Chain_Length, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Number_Of_Chains"), &HOST_Number_Of_Chains, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Table_Index"), &HOST_Table_Index, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("SHA1_Generate_Device_Number_Of_Threads"), &HOST_Number_Of_Threads, sizeof(HOST_Number_Of_Threads));
}


extern "C" void LaunchGenerateKernelSHA1(int passwordLength, uint32_t CUDA_Blocks,
        uint32_t CUDA_Threads, unsigned char *DEVICE_Initial_Passwords,
        unsigned char *DEVICE_End_Hashes, uint32_t PasswordSpaceOffset,
        uint32_t CurrentChainStartOffset, uint32_t StepsPerInvocation, uint32_t CharsetOffset) {
    switch (passwordLength) {
            case 6:
                MakeSHA1ChainLen6 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 7:
                MakeSHA1ChainLen7 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 8:
                MakeSHA1ChainLen8 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 9:
                MakeSHA1ChainLen9 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 10:
                MakeSHA1ChainLen10 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            default:
                printf("Password length %d not supported!", passwordLength);
                exit(1);
        }
}