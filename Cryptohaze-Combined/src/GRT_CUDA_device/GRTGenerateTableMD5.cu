#include "hip/hip_runtime.h"
/*
Cryptohaze GPU Rainbow Tables
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

// CUDA MD5 kernels for table generation.

// This is here so Netbeans doesn't error-spam my IDE
#if !defined(__HIPCC__)
    // define the keywords, so that the IDE does not complain about them
    #define __global__
    #define __device__
    #define __shared__
    #define __constant__
    #define blockIdx.x 1
    #define blockDim.x 1
    #define threadIdx.x 1
#endif

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>

#ifdef _WIN32
#include "windows/stdint.h"
#else
#include <stdint.h>
#endif

// Some CUDA variables
__device__ __constant__ unsigned char MD5_Generate_Device_Charset_Constant[512]; // Constant space for charset
__device__ __constant__ uint32_t MD5_Generate_Device_Charset_Length; // Character set length
__device__ __constant__ uint32_t MD5_Generate_Device_Chain_Length; // May as well pull it from constant memory... faster.
__device__ __constant__ uint32_t MD5_Generate_Device_Number_Of_Chains; // Same, may as well be constant.
__device__ __constant__ uint32_t MD5_Generate_Device_Table_Index;
__device__ __constant__ uint32_t MD5_Generate_Device_Number_Of_Threads; // It needs this, and can't easily calculate it


#include "../../inc/CUDA_Common/CUDA_MD5.h"
#include "../../inc/CUDA_Common/Hash_Common.h"
#include "../../inc/GRT_CUDA_device/CUDA_Reduction_Functions.h"
#include "../../inc/GRT_CUDA_device/CUDA_Load_Store_Registers.h"

/*
__global__ void MakeMD5ChainLen10(unsigned char *InitialPasswordArray, unsigned char *OutputHashArray,
    uint32_t PasswordSpaceOffset, uint32_t StartChainIndex, uint32_t StepsToRun, uint32_t charset_offset) {

    // Needed variables for generation
    uint32_t CurrentStep, PassCount, password_index;

    // Hash variables
    uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15;
    uint32_t a,b,c,d;

    // Word-width access to the arrays
    uint32_t *InitialArray32;
    uint32_t *OutputArray32;
    // 32-bit accesses to the hash arrays
    InitialArray32 = (uint32_t *)InitialPasswordArray;
    OutputArray32 = (uint32_t *)OutputHashArray;


    __shared__ char charset[512];

    // Generic "copy charset to shared memory" function
    copySingleCharsetToShared(charset, Device_Charset_Constant);

    // Figure out which password we are working on.
    password_index = ((blockIdx.x*blockDim.x + threadIdx.x) + (PasswordSpaceOffset * Device_Number_Of_Threads));

    // Return if this thread is working on something beyond the end of the password space
    if (password_index >= Device_Number_Of_Chains) {
        return;
    }

    clearB0toB15(b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15);
    // Load b0/b1 out of memory
    b0 = (uint32_t)InitialArray32[0 * Device_Number_Of_Chains + password_index];
    b1 = (uint32_t)InitialArray32[1 * Device_Number_Of_Chains + password_index];
    b2 = (uint32_t)InitialArray32[2 * Device_Number_Of_Chains + password_index];

    for (PassCount = 0; PassCount < StepsToRun; PassCount++) {
        CurrentStep = PassCount + StartChainIndex;

        padMDHash(10, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15);
        CUDA_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d);
        reduceSingleCharsetNormal(b0, b1, b2, a, b, c, d, CurrentStep, charset, charset_offset, 10, Device_Table_Index);

        charset_offset++;
        if (charset_offset >= Device_Charset_Length) {
            charset_offset = 0;
        }
    }
    // Done with the number of steps we need to run

    // If we are done (or have somehow overflowed), store the result
    if (CurrentStep >= (Device_Chain_Length - 1)) {
        OutputArray32[0 * Device_Number_Of_Chains + password_index] = a;
        OutputArray32[1 * Device_Number_Of_Chains + password_index] = b;
        OutputArray32[2 * Device_Number_Of_Chains + password_index] = c;
        OutputArray32[3 * Device_Number_Of_Chains + password_index] = d;
    }
    // Else, store the b0/b1 values back to the initial array for the next loop
    else {
        InitialArray32[0 * Device_Number_Of_Chains + password_index] = b0;
        InitialArray32[1 * Device_Number_Of_Chains + password_index] = b1;
        InitialArray32[2 * Device_Number_Of_Chains + password_index] = b2;
    }
}
*/



#define CREATE_MD5_GEN_KERNEL(length) \
__global__ void MakeMD5ChainLen##length(unsigned char *InitialPasswordArray, unsigned char *OutputHashArray, \
    uint32_t PasswordSpaceOffset, uint32_t StartChainIndex, uint32_t StepsToRun, uint32_t charset_offset) { \
    const int pass_length = length; \
    uint32_t CurrentStep, PassCount, password_index; \
    uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
    uint32_t a,b,c,d; \
    uint32_t *InitialArray32; \
    uint32_t *OutputArray32; \
    InitialArray32 = (uint32_t *)InitialPasswordArray; \
    OutputArray32 = (uint32_t *)OutputHashArray; \
    __shared__ char charset[512]; \
    copySingleCharsetToShared(charset, MD5_Generate_Device_Charset_Constant); \
    password_index = ((blockIdx.x*blockDim.x + threadIdx.x) + (PasswordSpaceOffset * MD5_Generate_Device_Number_Of_Threads)); \
    if (password_index >= MD5_Generate_Device_Number_Of_Chains) { \
        return; \
    } \
    clearB0toB15(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15); \
    LoadMD5RegistersFromGlobalMemory(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15, \
        InitialArray32, MD5_Generate_Device_Number_Of_Chains, password_index, pass_length); \
    for (PassCount = 0; PassCount < StepsToRun; PassCount++) { \
        CurrentStep = PassCount + StartChainIndex; \
        padMDHash(pass_length, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
        CUDA_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d); \
        reduceSingleCharsetNormal(b0, b1, b2, a, b, c, d, CurrentStep, charset, charset_offset, pass_length, MD5_Generate_Device_Table_Index); \
        charset_offset++; \
        if (charset_offset >= MD5_Generate_Device_Charset_Length) { \
            charset_offset = 0; \
        } \
    } \
    if (CurrentStep >= (MD5_Generate_Device_Chain_Length - 1)) { \
        OutputArray32[0 * MD5_Generate_Device_Number_Of_Chains + password_index] = a; \
        OutputArray32[1 * MD5_Generate_Device_Number_Of_Chains + password_index] = b; \
        OutputArray32[2 * MD5_Generate_Device_Number_Of_Chains + password_index] = c; \
        OutputArray32[3 * MD5_Generate_Device_Number_Of_Chains + password_index] = d; \
    } \
    else { \
    SaveMD5RegistersIntoGlobalMemory(b0,b1,b2,b3,b4,b5,b6,b7,b8,b9,b10,b11,b12,b13,b14,b15, \
        InitialArray32, MD5_Generate_Device_Number_Of_Chains, password_index, pass_length); \
    } \
}




CREATE_MD5_GEN_KERNEL(6)
CREATE_MD5_GEN_KERNEL(7)
CREATE_MD5_GEN_KERNEL(8)
CREATE_MD5_GEN_KERNEL(9)
CREATE_MD5_GEN_KERNEL(10)




extern "C" void copyConstantsToMD5(unsigned char *HOST_Charset, uint32_t HOST_Charset_Length,
    uint32_t HOST_Chain_Length, uint32_t HOST_Number_Of_Chains, uint32_t HOST_Table_Index,
    uint32_t HOST_Number_Of_Threads) {

    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Charset_Constant"),HOST_Charset, 512);
    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Charset_Length"), &HOST_Charset_Length, sizeof(uint32_t));

    // Copy general table parameters to constant space
    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Chain_Length"), &HOST_Chain_Length, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Number_Of_Chains"), &HOST_Number_Of_Chains, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Table_Index"), &HOST_Table_Index, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL("MD5_Generate_Device_Number_Of_Threads"), &HOST_Number_Of_Threads, sizeof(HOST_Number_Of_Threads));
}


extern "C" void LaunchGenerateKernelMD5(int passwordLength, uint32_t CUDA_Blocks,
        uint32_t CUDA_Threads, unsigned char *DEVICE_Initial_Passwords,
        unsigned char *DEVICE_End_Hashes, uint32_t PasswordSpaceOffset,
        uint32_t CurrentChainStartOffset, uint32_t StepsPerInvocation, uint32_t CharsetOffset) {
    switch (passwordLength) {
            case 6:
                MakeMD5ChainLen6 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 7:
                MakeMD5ChainLen7 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 8:
                MakeMD5ChainLen8 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 9:
                MakeMD5ChainLen9 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            case 10:
                MakeMD5ChainLen10 <<< CUDA_Blocks, CUDA_Threads >>>
                    (DEVICE_Initial_Passwords, DEVICE_End_Hashes, PasswordSpaceOffset,
                    CurrentChainStartOffset, StepsPerInvocation, CharsetOffset);
                break;
            default:
                printf("Password length %d not supported!", passwordLength);
                exit(1);
        }
}