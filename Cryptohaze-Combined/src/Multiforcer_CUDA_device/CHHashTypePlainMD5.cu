#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/


#include "Multiforcer_Common/CHCommon.h"

extern struct global_commands global_interface;



typedef uint32_t UINT4;
__device__ __constant__ char deviceCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN];
__device__ __constant__ __align__(16) unsigned char charsetLengths[MAX_PASSWORD_LEN];
__device__ __constant__ unsigned char constantBitmap[8192]; // for lookups


#include "Multiforcer_CUDA_device/CUDAcommon.h"
#include "CUDA_Common/CUDAMD5.h"


/*
#define MD5_CUDA_KERNEL_CREATE(length) \
__global__ void CUDA_MD5_Search_##length (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count,  \
				unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8)  unsigned char sharedLengths[MAX_PASSWORD_LEN]; \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositions(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15); \
  while (password_count < count) { \
  initMD(pass_length, sharedCharset, \
  	p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
	b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15);	 \
  CUDA_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d); \
  checkHashMulti(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
		a, b, c, d, b0, b1, b2, b3, b4, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}*/



// This is actually just as fast for small sizes, so we use it.
// Compiler optimizations FTW!
#define MD5_CUDA_KERNEL_CREATE_LONG(length) \
__global__ void CUDA_MD5_Search_##length (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count,  \
				unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
           p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
           p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
           p44, p45, p46, p47; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8)  unsigned char sharedLengths[MAX_PASSWORD_LEN]; \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositionsLong(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                   p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                   p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                   p44, p45, p46, p47); \
  while (password_count < count) { \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadPasswordAtPositionLong(pass_length, 0, sharedCharset, \
        p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
        p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
        p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
        p44, p45, p46, p47, \
        b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  CUDA_GENERIC_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, \
        a, b, c, d, pass_length); \
  checkHashMultiLong(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                p44, p45, p46, p47, \
		a, b, c, d, b0, b1, b2, b3, b4, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}

MD5_CUDA_KERNEL_CREATE_LONG(1)
MD5_CUDA_KERNEL_CREATE_LONG(2)
MD5_CUDA_KERNEL_CREATE_LONG(3)
MD5_CUDA_KERNEL_CREATE_LONG(4)
MD5_CUDA_KERNEL_CREATE_LONG(5)
MD5_CUDA_KERNEL_CREATE_LONG(6)
MD5_CUDA_KERNEL_CREATE_LONG(7)
MD5_CUDA_KERNEL_CREATE_LONG(8)
MD5_CUDA_KERNEL_CREATE_LONG(9)
MD5_CUDA_KERNEL_CREATE_LONG(10)
MD5_CUDA_KERNEL_CREATE_LONG(11)
MD5_CUDA_KERNEL_CREATE_LONG(12)
MD5_CUDA_KERNEL_CREATE_LONG(13)
MD5_CUDA_KERNEL_CREATE_LONG(14)
MD5_CUDA_KERNEL_CREATE_LONG(15)
MD5_CUDA_KERNEL_CREATE_LONG(16)
MD5_CUDA_KERNEL_CREATE_LONG(17)
MD5_CUDA_KERNEL_CREATE_LONG(18)
MD5_CUDA_KERNEL_CREATE_LONG(19)
MD5_CUDA_KERNEL_CREATE_LONG(20)
MD5_CUDA_KERNEL_CREATE_LONG(21)
MD5_CUDA_KERNEL_CREATE_LONG(22)
MD5_CUDA_KERNEL_CREATE_LONG(23)
MD5_CUDA_KERNEL_CREATE_LONG(24)
MD5_CUDA_KERNEL_CREATE_LONG(25)
MD5_CUDA_KERNEL_CREATE_LONG(26)
MD5_CUDA_KERNEL_CREATE_LONG(27)
MD5_CUDA_KERNEL_CREATE_LONG(28)
MD5_CUDA_KERNEL_CREATE_LONG(29)
MD5_CUDA_KERNEL_CREATE_LONG(30)
MD5_CUDA_KERNEL_CREATE_LONG(31)
MD5_CUDA_KERNEL_CREATE_LONG(32)
MD5_CUDA_KERNEL_CREATE_LONG(33)
MD5_CUDA_KERNEL_CREATE_LONG(34)
MD5_CUDA_KERNEL_CREATE_LONG(35)
MD5_CUDA_KERNEL_CREATE_LONG(36)
MD5_CUDA_KERNEL_CREATE_LONG(37)
MD5_CUDA_KERNEL_CREATE_LONG(38)
MD5_CUDA_KERNEL_CREATE_LONG(39)
MD5_CUDA_KERNEL_CREATE_LONG(40)
MD5_CUDA_KERNEL_CREATE_LONG(41)
MD5_CUDA_KERNEL_CREATE_LONG(42)
MD5_CUDA_KERNEL_CREATE_LONG(43)
MD5_CUDA_KERNEL_CREATE_LONG(44)
MD5_CUDA_KERNEL_CREATE_LONG(45)
MD5_CUDA_KERNEL_CREATE_LONG(46)
MD5_CUDA_KERNEL_CREATE_LONG(47)
MD5_CUDA_KERNEL_CREATE_LONG(48)

// Copy the shared variables to the host
extern "C" void copyMD5DataToConstant(char *hostCharset, int charsetLength,
        unsigned char *hostCharsetLengths, unsigned char *hostSharedBitmap, int threadId) {
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceCharset), hostCharset, (MAX_CHARSET_LENGTH * charsetLength)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantBitmap), hostSharedBitmap, 8192));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(charsetLengths), hostCharsetLengths, MAX_PASSWORD_LEN));
}

extern "C" void Launch_CUDA_MD5_Kernel(int passlength, uint64_t charsetLength, int numberOfPasswords, unsigned char *DEVICE_Passwords,
						unsigned char *DEVICE_Success, struct start_positions *DEVICE_Start_Positions, uint64_t per_step, uint64_t threads, uint64_t blocks, unsigned char *DEVICE_Hashes, unsigned char *DEVICE_Bitmap) {
    
    if (passlength == 1) {
	  CUDA_MD5_Search_1 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 2) {
	  CUDA_MD5_Search_2 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 3) {
	  CUDA_MD5_Search_3 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 4) {
	  CUDA_MD5_Search_4 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 5) {
	  CUDA_MD5_Search_5 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 6) {
	  CUDA_MD5_Search_6 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 7) {
	  CUDA_MD5_Search_7 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 8) {
	  CUDA_MD5_Search_8 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 9) {
	  CUDA_MD5_Search_9 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 10) {
	  CUDA_MD5_Search_10 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 11) {
	  CUDA_MD5_Search_11 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 12) {
	  CUDA_MD5_Search_12 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 13) {
	  CUDA_MD5_Search_13 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 14) {
	  CUDA_MD5_Search_14 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 15) {
	  CUDA_MD5_Search_15 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 16) {
	  CUDA_MD5_Search_16 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 17) {
          CUDA_MD5_Search_17 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 18) {
          CUDA_MD5_Search_18 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 19) {
          CUDA_MD5_Search_19 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 20) {
          CUDA_MD5_Search_20 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 21) {
          CUDA_MD5_Search_21 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 22) {
          CUDA_MD5_Search_22 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 23) {
          CUDA_MD5_Search_23 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 24) {
          CUDA_MD5_Search_24 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 25) {
          CUDA_MD5_Search_25 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 26) {
          CUDA_MD5_Search_26 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 27) {
          CUDA_MD5_Search_27 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 28) {
          CUDA_MD5_Search_28 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 29) {
          CUDA_MD5_Search_29 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 30) {
          CUDA_MD5_Search_30 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 31) {
          CUDA_MD5_Search_31 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 32) {
          CUDA_MD5_Search_32 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 33) {
          CUDA_MD5_Search_33 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 34) {
          CUDA_MD5_Search_34 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 35) {
          CUDA_MD5_Search_35 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 36) {
          CUDA_MD5_Search_36 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 37) {
          CUDA_MD5_Search_37 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 38) {
          CUDA_MD5_Search_38 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 39) {
          CUDA_MD5_Search_39 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 40) {
          CUDA_MD5_Search_40 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 41) {
          CUDA_MD5_Search_41 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 42) {
          CUDA_MD5_Search_42 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 43) {
          CUDA_MD5_Search_43 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 44) {
          CUDA_MD5_Search_44 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 45) {
          CUDA_MD5_Search_45 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 46) {
          CUDA_MD5_Search_46 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 47) {
          CUDA_MD5_Search_47 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 48) {
	  CUDA_MD5_Search_48 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else {
            sprintf(global_interface.exit_message, "MD5 length >48 not currently supported!\n");
            global_interface.exit = 1;
            return;
        }

        hipError_t err = hipGetLastError();
    if( hipSuccess != err)
      {
        sprintf(global_interface.exit_message, "Cuda error: %s.\n", hipGetErrorString( err) );
        global_interface.exit = 1;
        return;
      }
}