#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/


#include "Multiforcer_Common/CHCommon.h"


extern struct global_commands global_interface;


typedef uint32_t UINT4;
__device__ __constant__ char deviceCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN];
__device__ __constant__ __align__(16) unsigned char charsetLengths[MAX_PASSWORD_LEN];
__device__ __constant__ unsigned char constantBitmap[8192]; // for lookups


#include "Multiforcer_CUDA_device/CUDAcommon.h"

#include "CUDA_Common/CUDASHA1.h"



#define CUDA_SHA1_KERNEL_CREATE(length) \
__global__ void CUDA_SHA1_Search_##length (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count, unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d, e; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8) unsigned char sharedLengths[16];  \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositions(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15); \
  while (password_count < count) { \
  initSHA1(pass_length, sharedCharset, \
  	p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
	b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15);	 \
  SHA_TRANSFORM_SMALL(a, b, c, d, e, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  checkHashMultiSHA1(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
		b0, b1, b2, b3, b4, a, b, c, d, e, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}



#define CUDA_SHA1_KERNEL_CREATELONG(length) \
__global__ void CUDA_SHA1_Search_##length (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count, unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d, e; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
           p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
           p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
           p44, p45, p46, p47; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8) unsigned char sharedLengths[MAX_PASSWORD_LEN];  \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositionsLong(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                   p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                   p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                   p44, p45, p46, p47); \
  while (password_count < count) { \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadPasswordAtPositionLong(pass_length, 0, sharedCharset, \
        p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
        p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
        p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
        p44, p45, p46, p47, \
        b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  b15 = ((pass_length * 8) & 0xff) << 24 | (((pass_length * 8) >> 8) & 0xff) << 16; \
  SetCharacterAtPosition(0x80, pass_length, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15 ); \
  SHA_TRANSFORM(a, b, c, d, e, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  checkHashMultiSHA1Long(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                p44, p45, p46, p47, \
		b0, b1, b2, b3, b4, a, b, c, d, e, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}


CUDA_SHA1_KERNEL_CREATE(1)
CUDA_SHA1_KERNEL_CREATE(2)
CUDA_SHA1_KERNEL_CREATE(3)
CUDA_SHA1_KERNEL_CREATE(4)
CUDA_SHA1_KERNEL_CREATE(5)
CUDA_SHA1_KERNEL_CREATE(6)
CUDA_SHA1_KERNEL_CREATE(7)
CUDA_SHA1_KERNEL_CREATE(8)
CUDA_SHA1_KERNEL_CREATE(9)
CUDA_SHA1_KERNEL_CREATE(10)
// Can be short above this...
CUDA_SHA1_KERNEL_CREATELONG(11)
CUDA_SHA1_KERNEL_CREATELONG(12)
CUDA_SHA1_KERNEL_CREATELONG(13)
CUDA_SHA1_KERNEL_CREATELONG(14)
CUDA_SHA1_KERNEL_CREATELONG(15)
CUDA_SHA1_KERNEL_CREATELONG(16)
CUDA_SHA1_KERNEL_CREATELONG(17)
CUDA_SHA1_KERNEL_CREATELONG(18)
CUDA_SHA1_KERNEL_CREATELONG(19)
CUDA_SHA1_KERNEL_CREATELONG(20)
CUDA_SHA1_KERNEL_CREATELONG(21)
CUDA_SHA1_KERNEL_CREATELONG(22)
CUDA_SHA1_KERNEL_CREATELONG(23)
CUDA_SHA1_KERNEL_CREATELONG(24)
CUDA_SHA1_KERNEL_CREATELONG(25)
CUDA_SHA1_KERNEL_CREATELONG(26)
CUDA_SHA1_KERNEL_CREATELONG(27)
CUDA_SHA1_KERNEL_CREATELONG(28)
CUDA_SHA1_KERNEL_CREATELONG(29)
CUDA_SHA1_KERNEL_CREATELONG(30)
CUDA_SHA1_KERNEL_CREATELONG(31)
CUDA_SHA1_KERNEL_CREATELONG(32)
CUDA_SHA1_KERNEL_CREATELONG(33)
CUDA_SHA1_KERNEL_CREATELONG(34)
CUDA_SHA1_KERNEL_CREATELONG(35)
CUDA_SHA1_KERNEL_CREATELONG(36)
CUDA_SHA1_KERNEL_CREATELONG(37)
CUDA_SHA1_KERNEL_CREATELONG(38)
CUDA_SHA1_KERNEL_CREATELONG(39)
CUDA_SHA1_KERNEL_CREATELONG(40)
CUDA_SHA1_KERNEL_CREATELONG(41)
CUDA_SHA1_KERNEL_CREATELONG(42)
CUDA_SHA1_KERNEL_CREATELONG(43)
CUDA_SHA1_KERNEL_CREATELONG(44)
CUDA_SHA1_KERNEL_CREATELONG(45)
CUDA_SHA1_KERNEL_CREATELONG(46)
CUDA_SHA1_KERNEL_CREATELONG(47)
CUDA_SHA1_KERNEL_CREATELONG(48)

// Copy the shared variables to the host
extern "C" void copySHA1DataToConstant(char *hostCharset, int charsetLength,
        unsigned char *hostCharsetLengths, unsigned char *hostSharedBitmap, int threadId) {
    //printf("Thread %d in CHHashTypeNTLM.cu, copyNTLMDataToCharset()\n", threadId);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceCharset), hostCharset, (MAX_CHARSET_LENGTH * charsetLength)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantBitmap), hostSharedBitmap, 8192));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(charsetLengths), hostCharsetLengths, 16));
}



extern "C" void Launch_CUDA_SHA1_Kernel(int passlength, uint64_t charsetLength, int numberOfPasswords, unsigned char *DEVICE_Passwords,
        unsigned char *DEVICE_Success, struct start_positions *DEVICE_Start_Positions, uint64_t per_step, uint64_t threads, uint64_t blocks, unsigned char *DEVICE_Hashes, unsigned char *DEVICE_Bitmap) {
    if (passlength == 1) {
	  CUDA_SHA1_Search_1 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 2) {
	  CUDA_SHA1_Search_2 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 3) {
	  CUDA_SHA1_Search_3 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 4) {
	  CUDA_SHA1_Search_4 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 5) {
	  CUDA_SHA1_Search_5 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 6) {
	  CUDA_SHA1_Search_6 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 7) {
	  CUDA_SHA1_Search_7 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 8) {
	  CUDA_SHA1_Search_8 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 9) {
	  CUDA_SHA1_Search_9 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 10) {
	  CUDA_SHA1_Search_10 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 11) {
	  CUDA_SHA1_Search_11 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 12) {
	  CUDA_SHA1_Search_12 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 13) {
	  CUDA_SHA1_Search_13 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 14) {
	  CUDA_SHA1_Search_14 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 15) {
	  CUDA_SHA1_Search_15 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 16) {
	  CUDA_SHA1_Search_16 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 17) {
          CUDA_SHA1_Search_17 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 18) {
          CUDA_SHA1_Search_18 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 19) {
          CUDA_SHA1_Search_19 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 20) {
          CUDA_SHA1_Search_20 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 21) {
          CUDA_SHA1_Search_21 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 22) {
          CUDA_SHA1_Search_22 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 23) {
          CUDA_SHA1_Search_23 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 24) {
          CUDA_SHA1_Search_24 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 25) {
          CUDA_SHA1_Search_25 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 26) {
          CUDA_SHA1_Search_26 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 27) {
          CUDA_SHA1_Search_27 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 28) {
          CUDA_SHA1_Search_28 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 29) {
          CUDA_SHA1_Search_29 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 30) {
          CUDA_SHA1_Search_30 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 31) {
          CUDA_SHA1_Search_31 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 32) {
          CUDA_SHA1_Search_32 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 33) {
          CUDA_SHA1_Search_33 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 34) {
          CUDA_SHA1_Search_34 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 35) {
          CUDA_SHA1_Search_35 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 36) {
          CUDA_SHA1_Search_36 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 37) {
          CUDA_SHA1_Search_37 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 38) {
          CUDA_SHA1_Search_38 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 39) {
          CUDA_SHA1_Search_39 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 40) {
          CUDA_SHA1_Search_40 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 41) {
          CUDA_SHA1_Search_41 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 42) {
          CUDA_SHA1_Search_42 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 43) {
          CUDA_SHA1_Search_43 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 44) {
          CUDA_SHA1_Search_44 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 45) {
          CUDA_SHA1_Search_45 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 46) {
          CUDA_SHA1_Search_46 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 47) {
          CUDA_SHA1_Search_47 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 48) {
          CUDA_SHA1_Search_48 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else {
            sprintf(global_interface.exit_message, "SHA1 length >48 not currently supported!\n");
            global_interface.exit = 1;
            return;
        }
	hipError_t err = hipGetLastError();
    if( hipSuccess != err)
      {
        sprintf(global_interface.exit_message, "Cuda error: %s.\n", hipGetErrorString( err) );
        global_interface.exit = 1;
        return;
      }
}