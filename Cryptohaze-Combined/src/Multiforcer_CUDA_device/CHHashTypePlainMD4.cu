#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/


#include "Multiforcer_Common/CHCommon.h"

extern struct global_commands global_interface;


typedef uint32_t UINT4;
__device__ __constant__ char deviceCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN];
__device__ __constant__ __align__(16) unsigned char charsetLengths[16];
__device__ __constant__ unsigned char constantBitmap[8192]; // for lookups


#include "Multiforcer_CUDA_device/CUDAcommon.h"
#include "CUDA_Common/CUDAMD4.h"


#define MD4_CUDA_KERNEL_CREATE(length) \
__global__ void CUDA_MD4_Search_##length (unsigned char *OutputPassword, unsigned char *success,  \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, \
				unsigned int count, unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ unsigned char sharedLengths[16]; \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositions(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15); \
  while (password_count < count) { \
  initMD(pass_length, sharedCharset, \
  	p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
	b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15);	 \
  CUDA_MD4(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, a, b, c, d); \
  checkHashMulti(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
		a, b, c, d, b0, b1, b2, b3, b4, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}


MD4_CUDA_KERNEL_CREATE(1)
MD4_CUDA_KERNEL_CREATE(2)
MD4_CUDA_KERNEL_CREATE(3)
MD4_CUDA_KERNEL_CREATE(4)
MD4_CUDA_KERNEL_CREATE(5)
MD4_CUDA_KERNEL_CREATE(6)
MD4_CUDA_KERNEL_CREATE(7)
MD4_CUDA_KERNEL_CREATE(8)
MD4_CUDA_KERNEL_CREATE(9)
MD4_CUDA_KERNEL_CREATE(10)
MD4_CUDA_KERNEL_CREATE(11)
MD4_CUDA_KERNEL_CREATE(12)
MD4_CUDA_KERNEL_CREATE(13)
MD4_CUDA_KERNEL_CREATE(14)
MD4_CUDA_KERNEL_CREATE(15)
MD4_CUDA_KERNEL_CREATE(16)

// Copy the shared variables to the host
extern "C" void copyMD4DataToConstant(char *hostCharset, int charsetLength,
        unsigned char *hostCharsetLengths, unsigned char *hostSharedBitmap, int threadId) {
    //printf("Thread %d in CHHashTypeMD5.cu, copyMD5DataToCharset()\n", threadId);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceCharset), hostCharset, (MAX_CHARSET_LENGTH * charsetLength)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantBitmap), hostSharedBitmap, 8192));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(charsetLengths), hostCharsetLengths, 16));
}

extern "C" void Launch_CUDA_MD4_Kernel(int passlength, uint64_t charsetLength, int numberOfPasswords, unsigned char *DEVICE_Passwords,
						unsigned char *DEVICE_Success, struct start_positions *DEVICE_Start_Positions, uint64_t per_step, uint64_t threads, uint64_t blocks, unsigned char *DEVICE_Hashes, unsigned char *DEVICE_Bitmap) {
    if (passlength == 1) {
	  CUDA_MD4_Search_1 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 2) {
	  CUDA_MD4_Search_2 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 3) {
	  CUDA_MD4_Search_3 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 4) {
	  CUDA_MD4_Search_4 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 5) {
	  CUDA_MD4_Search_5 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 6) {
	  CUDA_MD4_Search_6 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 7) {
	  CUDA_MD4_Search_7 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 8) {
	  CUDA_MD4_Search_8 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 9) {
	  CUDA_MD4_Search_9 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 10) {
	  CUDA_MD4_Search_10 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 11) {
	  CUDA_MD4_Search_11 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 12) {
	  CUDA_MD4_Search_12 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 13) {
	  CUDA_MD4_Search_13 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 14) {
	  CUDA_MD4_Search_14 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 15) {
	  CUDA_MD4_Search_15 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 16) {
	  CUDA_MD4_Search_16 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	}

	hipError_t err = hipGetLastError();
    if( hipSuccess != err)
      {
        sprintf(global_interface.exit_message, "Cuda error: %s.\n", hipGetErrorString( err) );
        global_interface.exit = 1;
        return;
      }
}