#include "hip/hip_runtime.h"
/*
Cryptohaze Multiforcer & Wordyforcer - low performance GPU password cracking
Copyright (C) 2011  Bitweasil (http://www.cryptohaze.com/)

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include "Multiforcer_Common/CHCommon.h"

extern struct global_commands global_interface;



typedef uint32_t UINT4;
__device__ __constant__ char deviceCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN];
__device__ __constant__ __align__(16) unsigned char charsetLengths[MAX_PASSWORD_LEN];
__device__ __constant__ unsigned char constantBitmap[8192]; // for lookups



#include "Multiforcer_CUDA_device/CUDAcommon.h"
#include "CUDA_Common/CUDAMD5.h"
#include "CUDA_Common/CUDASHA1.h"




/*
__global__ void CUDA_SHA1OfMD5_Search_6 (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count,  \
				unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = 6; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d,e; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
           p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
           p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
           p44, p45, p46, p47; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8)  unsigned char sharedLengths[MAX_PASSWORD_LEN]; \
  __shared__               char hashLookup[256][2]; \
  loadHashLookup(hashLookup); \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositionsLong(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                   p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                   p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                   p44, p45, p46, p47); \
  while (password_count < count) { \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadPasswordAtPositionLong(pass_length, 0, sharedCharset, \
        p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
        p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
        p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
        p44, p45, p46, p47, \
        b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  CUDA_GENERIC_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, \
        a, b, c, d, pass_length); \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadHashAsString(hashLookup, a, b, c, d, b0, b1, b2, b3, b4, b5, b6, b7); \
  b15 = ((32 * 8) & 0xff) << 24 | (((32 * 8) >> 8) & 0xff) << 16; \
  SetCharacterAtPosition(0x80, 32, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15 ); \
  SHA_TRANSFORM(a, b, c, d, e, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  checkHashMultiSHA1Long(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                p44, p45, p46, p47, \
		b0, b1, b2, b3, b4, a, b, c, d, e, b5); \
  password_count++; \
  incrementCounters6Multi(); \
  } \
}
*/

#define SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(length) \
__global__ void CUDA_SHA1OfMD5_Search_##length (unsigned char *OutputPassword, unsigned char *success, \
			    int charsetLen, uint32_t numberOfPasswords, struct start_positions *DEVICE_Start_Positions, unsigned int count,  \
				unsigned char * DEVICE_Hashes, unsigned char *DEVICE_HashTable) { \
  const int pass_length = length; \
  uint32_t b0,b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15; \
  uint32_t a,b,c,d,e; \
  uint32_t thread_index = blockIdx.x*blockDim.x + threadIdx.x; \
  uint32_t *DEVICE_Hashes_32 = (uint32_t *)DEVICE_Hashes; \
  unsigned char p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
           p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
           p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
           p44, p45, p46, p47; \
  UINT4 password_count = 0; \
  __shared__ __align__(16) unsigned char sharedCharset[MAX_CHARSET_LENGTH * MAX_PASSWORD_LEN]; \
  __shared__ __align__(16) unsigned char sharedBitmap[8192]; \
  __shared__ __align__(8)  unsigned char sharedLengths[MAX_PASSWORD_LEN]; \
  __shared__               char hashLookup[256][2]; \
  loadHashLookup(hashLookup, values); \
  copyCharsetAndBitmap(sharedCharset, sharedBitmap, sharedLengths, charsetLen, pass_length); \
  loadStartPositionsLong(pass_length, thread_index, DEVICE_Start_Positions,  \
		   p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                   p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                   p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                   p44, p45, p46, p47); \
  while (password_count < count) { \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadPasswordAtPositionLong(pass_length, 0, sharedCharset, \
        p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
        p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
        p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
        p44, p45, p46, p47, \
        b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  CUDA_GENERIC_MD5(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, \
        a, b, c, d, pass_length); \
  clearB0toB15(b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  LoadHashAsString(hashLookup, a, b, c, d, b0, b1, b2, b3, b4, b5, b6, b7); \
  b15 = ((32 * 8) & 0xff) << 24 | (((32 * 8) >> 8) & 0xff) << 16; \
  SetCharacterAtPosition(0x80, 32, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15 ); \
  SHA_TRANSFORM(a, b, c, d, e, b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15); \
  checkHashMultiSHA1Long(pass_length, sharedBitmap, DEVICE_HashTable, numberOfPasswords, \
		DEVICE_Hashes_32, success, OutputPassword, \
		p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, \
                p16, p17, p18, p19, p20, p21, p22, p23, p24, p25, p26, p27, p28, p29, \
                p30, p31, p32, p33, p34, p35, p36, p37, p38, p39, p40, p41, p42, p43, \
                p44, p45, p46, p47, \
		b0, b1, b2, b3, b4, a, b, c, d, e, b5); \
  password_count++; \
  incrementCounters##length##Multi(); \
  } \
}

SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(1);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(2);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(3);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(4);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(5);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(6);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(7);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(8);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(9);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(10);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(11);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(12);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(13);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(14);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(15);
SHA1OFMD5_CUDA_KERNEL_CREATE_LONG(16);

// Copy the shared variables to the host
extern "C" void copySHA1OfMD5DataToConstant(char *hostCharset, int charsetLength,
        unsigned char *hostCharsetLengths, unsigned char *hostSharedBitmap, int threadId) {
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceCharset), hostCharset, (MAX_CHARSET_LENGTH * charsetLength)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constantBitmap), hostSharedBitmap, 8192));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(charsetLengths), hostCharsetLengths, MAX_PASSWORD_LEN));
}

extern "C" void Launch_CUDA_SHA1OfMD5_Kernel(int passlength, uint64_t charsetLength, int numberOfPasswords, unsigned char *DEVICE_Passwords,
						unsigned char *DEVICE_Success, struct start_positions *DEVICE_Start_Positions, uint64_t per_step, uint64_t threads, uint64_t blocks, unsigned char *DEVICE_Hashes, unsigned char *DEVICE_Bitmap) {

    if (passlength == 1) {
	  CUDA_SHA1OfMD5_Search_1 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 2) {
	  CUDA_SHA1OfMD5_Search_2 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 3) {
	  CUDA_SHA1OfMD5_Search_3 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 4) {
	  CUDA_SHA1OfMD5_Search_4 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 5) {
	  CUDA_SHA1OfMD5_Search_5 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 6) {
	  CUDA_SHA1OfMD5_Search_6 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 7) {
	  CUDA_SHA1OfMD5_Search_7 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 8) {
	  CUDA_SHA1OfMD5_Search_8 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 9) {
	  CUDA_SHA1OfMD5_Search_9 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 10) {
	  CUDA_SHA1OfMD5_Search_10 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 11) {
	  CUDA_SHA1OfMD5_Search_11 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 12) {
	  CUDA_SHA1OfMD5_Search_12 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 13) {
	  CUDA_SHA1OfMD5_Search_13 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 14) {
	  CUDA_SHA1OfMD5_Search_14 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 15) {
	  CUDA_SHA1OfMD5_Search_15 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} else if (passlength == 16) {
	  CUDA_SHA1OfMD5_Search_16 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } /*else if (passlength == 17) {
          CUDA_DoubleMD5_Search_17 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 18) {
          CUDA_DoubleMD5_Search_18 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 19) {
          CUDA_DoubleMD5_Search_19 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 20) {
          CUDA_DoubleMD5_Search_20 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 21) {
          CUDA_DoubleMD5_Search_21 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 22) {
          CUDA_DoubleMD5_Search_22 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 23) {
          CUDA_DoubleMD5_Search_23 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 24) {
          CUDA_DoubleMD5_Search_24 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 25) {
          CUDA_DoubleMD5_Search_25 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 26) {
          CUDA_DoubleMD5_Search_26 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 27) {
          CUDA_DoubleMD5_Search_27 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 28) {
          CUDA_DoubleMD5_Search_28 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 29) {
          CUDA_DoubleMD5_Search_29 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 30) {
          CUDA_DoubleMD5_Search_30 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 31) {
          CUDA_DoubleMD5_Search_31 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 32) {
          CUDA_DoubleMD5_Search_32 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 33) {
          CUDA_DoubleMD5_Search_33 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 34) {
          CUDA_DoubleMD5_Search_34 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 35) {
          CUDA_DoubleMD5_Search_35 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 36) {
          CUDA_DoubleMD5_Search_36 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 37) {
          CUDA_DoubleMD5_Search_37 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 38) {
          CUDA_DoubleMD5_Search_38 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 39) {
          CUDA_DoubleMD5_Search_39 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 40) {
          CUDA_DoubleMD5_Search_40 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 41) {
          CUDA_DoubleMD5_Search_41 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 42) {
          CUDA_DoubleMD5_Search_42 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 43) {
          CUDA_DoubleMD5_Search_43 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 44) {
          CUDA_DoubleMD5_Search_44 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 45) {
          CUDA_DoubleMD5_Search_45 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 46) {
          CUDA_DoubleMD5_Search_46 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 47) {
          CUDA_DoubleMD5_Search_47 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
        } else if (passlength == 48) {
	  CUDA_DoubleMD5_Search_48 <<< blocks, threads >>> (DEVICE_Passwords, DEVICE_Success, charsetLength, numberOfPasswords, DEVICE_Start_Positions, per_step, DEVICE_Hashes, DEVICE_Bitmap);
	} */else {
            sprintf(global_interface.exit_message, "MD5 of SHA1 length >48 not currently supported!\n");
            global_interface.exit = 1;
            return;
        }
        hipError_t err = hipGetLastError();
    if( hipSuccess != err)
      {
        sprintf(global_interface.exit_message, "Cuda error: %s.\n", hipGetErrorString( err) );
        global_interface.exit = 1;
        return;
      }
}